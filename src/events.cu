/***************************************************************************
*   Original Diku Mud copyright (C) 1990, 1991 by Sebastian Hammer,        *
*   Michael Seifert, Hans Henrik St�rfeldt, Tom Madsen, and Katja Nyboe.   *
*                                                                          *
*   Merc Diku Mud improvements copyright (C) 1992, 1993 by Michael         *
*   Chastain, Michael Quan, and Mitchell Tse.                              *
*                                                                          *
*       ROM 2.4 is copyright 1993-1995 Russ Taylor                         *
*       ROM has been brought to you by the ROM consortium                  *
*           Russ Taylor (rtaylor@pacinfo.com)                              *
*           Gabrielle Taylor (gtaylor@pacinfo.com)                         *
*           Brian Moore (rom@rom.efn.org)                                  *
*       ROT 2.0 is copyright 1996-1999 by Russ Walsh                       *
*       DRM 1.0a is copyright 2000-2002 by Joshua Chance Blackwell         *
*        SD 2.0 is copyright 2004-2006 by Patrick Mylund Nielsen           *
***************************************************************************/

#if defined(macintosh)
#include <types.h>
#else
#include <sys/types.h>
#endif
#include <stdio.h>
#include <string.h>
#include <time.h>
#include "merc.h"
#include "globals.h"
#include "proto.h"

#define WOLLA(fun)   void fun (void)

WOLLA ( dailyevents )
{ 
/* Basic events that have their own chance check */

      if ( its_christmas || special_day )
      {
         global_exp = TRUE;
         double_bones = TRUE;
         double_damage = TRUE;
         double_stance = TRUE;
         double_pracs = TRUE;
      }
      else
      {
        event_doublexp     (  );
        event_doublebones     (  );
        event_doubledamage (  );
        event_doublestance (  );
        event_doublepracs  (  );
      }
      
/* Random pill and angel/demon token spreading */
        random_spread (  );
      
/* Happy Moment */
    if ( !its_christmas )
    {
     if ( bigchance ( 3 ) )
	 happymoment (  );
    }
    else
    {
     if ( chance ( 4 ) )
         merrychristmas (  );
    }

/* Auto Freezetag Game */
    if ( bigchance ( 3 ) )
	auto_tag (  );

/* Stock Market Crash */
    if ( bigchance ( 1 ) )
	event_marketcrash (  );
}

WOLLA ( event_doublexp )
{
    if ( (!global_exp) && ( bigchance ( 2 ) ) ) 
    { 
	global_exp = TRUE;
	do_gmessage ( "{R[{WEvent{R] {CIt is time for doublexp! Get killing!{x\n\r" );
	return;
    }
    else if ( (global_exp) && ( chance ( 25 ) ) ) 
    { 
	global_exp = FALSE;
	do_gmessage ( "{R[{WEvent{R] {CDoublexp is over.{x\n\r" );
	return;
    }
    return;
} 

WOLLA ( event_doublebones )
{
    if ( (!double_bones) && ( bigchance ( 2 ) ) ) 
    { 
	double_bones = TRUE;
	do_gmessage ( "{R[{WEvent{R] {CIt is time for double bones!{x\n\r" );
	return;
    }
    else if ( (double_bones) && ( chance ( 25 ) ) ) 
    { 
	double_bones = FALSE;
	do_gmessage ( "{R[{WEvent{R] {CDouble bones is over.{x\n\r" );
	return;
    }
    return;
} 

WOLLA ( event_doubledamage )
{
    if ( (!double_damage) && ( bigchance ( 2 ) ) ) 
    { 
	double_damage = TRUE;
	do_gmessage ( "{R[{WEvent{R] {CIt is time for double damage! Get smashing!{x\n\r" );
	return;
    }
    else if ( (double_damage) && ( chance ( 25 ) ) ) 
    { 
	double_damage = FALSE;
	do_gmessage ( "{R[{WEvent{R] {CDouble damage is over.{x\n\r" );
	return;
    }
    return;
} 

WOLLA ( event_doublestance )
{
    if ( (!double_stance) && ( bigchance ( 2 ) ) ) 
    { 
	double_stance = TRUE;
	do_gmessage ( "{R[{WEvent{R] {CIt is time for double stance gain! Get fighting!{x\n\r" );
	return;
    }
    else if ( (double_stance) && ( chance ( 25 ) ) ) 
    { 
	double_stance = FALSE;
	do_gmessage ( "{R[{WEvent{R] {CDouble stance gain is over.{x\n\r" );
	return;
    }
    return;
} 

WOLLA ( event_doublepracs )
{
    if ( (!double_pracs) && ( bigchance ( 2 ) ) ) 
    { 
	double_pracs = TRUE;
	do_gmessage ( "{R[{WEvent{R] {CIt is time for double practice gain!{x\n\r" );
	return;
    }
    else if ( (double_pracs) && ( chance ( 25 ) ) ) 
    { 
	double_pracs = FALSE;
	do_gmessage ( "{R[{WEvent{R] {CDouble practice gain is over.{x\n\r" );
	return;
    }
    return;
} 

WOLLA ( happymoment )
{
    CHAR_DATA *ch;
    DESCRIPTOR_DATA *d;
    long  happymomentbonus;

    for ( d = descriptor_list; d != NULL; d = d->next )
    {
     if ( d->connected == CON_PLAYING && ( ch = ( d->original ? d->original : d->character ) ) != NULL )
     {
    send_to_char ( "{*I{*t{*'{*s {*H{*a{*p{*p{*y {*M{*o{*m{*e{*n{*t{*!{*!{*!{x\n\r", d->character );
    send_to_char ( "{*I{*t{*'{*s {*H{*a{*p{*p{*y {*M{*o{*m{*e{*n{*t{*!{*!{*!{x\n\r", d->character );
    send_to_char ( "{*I{*t{*'{*s {*H{*a{*p{*p{*y {*M{*o{*m{*e{*n{*t{*!{*!{*!{x\n\r", d->character );
    send_to_char ( "{*I{*t{*'{*s {*H{*a{*p{*p{*y {*M{*o{*m{*e{*n{*t{*!{*!{*!{x\n\r", d->character );
    send_to_char ( "{*I{*t{*'{*s {*H{*a{*p{*p{*y {*M{*o{*m{*e{*n{*t{*!{*!{*!{x\n\r", d->character );
    send_to_char ( "{*I{*t{*'{*s {*H{*a{*p{*p{*y {*M{*o{*m{*e{*n{*t{*!{*!{*!{x\n\r\n\r", d->character );

     if ( ( d->character->level < HERO ) && !IS_SET ( d->character->act2, PLR2_NO_EXP ) )
     {
       if ( chance ( 70 ) )
       {
         happymomentbonus = number_range ( 100, 10000 );
         gain_exp ( d->character, happymomentbonus );
         printf_to_char ( d->character, "Free {Yexp{x!!! {r({R%ld{r){x (If you want to avoid this, type noexp)\n\r", happymomentbonus );
       }
       else send_to_char ( "Sorry, you didn't get any {Yexp{x this time.\n\r", d->character );
      }

    if ( chance ( 60 ) )
    {
     happymomentbonus = number_range ( 1, 300 );
     d->character->platinum += happymomentbonus;
     printf_to_char ( d->character, "Your {Yplatinum{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Yplatinum{x this time.\n\r", d->character );

    if ( chance ( 30 ) )
    {
     happymomentbonus = number_range ( 1, 5 );
     d->character->bones += happymomentbonus;
     printf_to_char ( d->character, "Your {Yiqp{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Yiqps{x this time.\n\r", d->character );

    if ( chance ( 40 ) )
    {
     happymomentbonus = number_range ( 10, 1000 );
     d->character->bones += happymomentbonus;
     printf_to_char ( d->character, "Your {Yaqp{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Yaqps{x this time.\n\r", d->character );

    if ( chance ( 10 ) )
    {
     happymomentbonus = number_range ( 1, 100 );
     d->character->max_hit += happymomentbonus;
     printf_to_char ( d->character, "Your {Yhp{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Yhps{x this time.\n\r", d->character );

    if ( chance ( 12 ) )
    {
     happymomentbonus = number_range ( 1, 200 );
     d->character->max_mana += happymomentbonus;
     printf_to_char ( d->character, "Your {Ymana{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Ymana{x this time.\n\r", d->character );

    if ( chance ( 18 ) )
    {
     happymomentbonus = number_range ( 1, 300 );
     d->character->max_move += happymomentbonus;
     printf_to_char ( d->character, "Your {Ymove{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Ymoves{x this time.\n\r", d->character );

    if ( chance ( 35 ) )
    {
     happymomentbonus = number_range ( 1, 50 );
     d->character->practice += happymomentbonus;
     printf_to_char ( d->character, "Your {Ypractice{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Ypractices{x this time.\n\r", d->character );

    if ( chance ( 35 ) )
    {
     happymomentbonus = number_range ( 1, 10 );
     d->character->train += happymomentbonus;
     printf_to_char ( d->character, "Your {Ytrain{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Ytrains{x this time.\n\r", d->character );

    d->character->hit = d->character->max_hit;
    d->character->mana = d->character->max_mana;
    d->character->move = d->character->max_move;
    send_to_char ( "{*The gods have restored you!{x\n\r", d->character );

     if ( d->character->level >= 35 )
     {
        save_char_obj ( d->character );
        send_to_char ( "Your character has been backupped. Please remember to do this frequently.\n\r", d->character );
     }
  }
 }
}

WOLLA ( merrychristmas )
{
    CHAR_DATA *ch;
    DESCRIPTOR_DATA *d;
    long  happymomentbonus;

    for ( d = descriptor_list; d != NULL; d = d->next )
    {
     if ( d->connected == CON_PLAYING && ( ch = ( d->original ? d->original : d->character ) ) != NULL )
     {
    send_to_char ( "{W   _ __ __                             __\n\r", d->character );
    send_to_char ( "  / )  )  )                          /   ) /                 /\n\r", d->character );
    send_to_char ( "   /  /  /   _   o_   o_   _       /      /         o    ---/---\n\r", d->character );
    send_to_char ( "  /  /  /   /_) /  ) /  ) / )  /  |      /_   o_           /  _ _   __\n\r", d->character );
    send_to_char ( " /  /   )__(___/  (_/  (_/ (__/   |     /  ) /  ) /  /)   /  / ) ) /  |  /)\n\r", d->character );
    send_to_char ( "                          ___/_    )___/  (_/  (_/(_/__)_/(_/ / (_/)_/(_/__){x\n\r", d->character );

     if ( ( d->character->level < HERO ) && !IS_SET ( d->character->act2, PLR2_NO_EXP ) )
     {
       if ( chance ( 70 ) )
       {
         happymomentbonus = number_range ( 100, 20000 );
         gain_exp ( d->character, happymomentbonus );
         printf_to_char ( d->character, "Free {Yexp{x!!! {r({R%ld{r){x (If you want to avoid this, type noexp)\n\r", happymomentbonus );
       }
       else send_to_char ( "Sorry, you didn't get any {Yexp{x this time.\n\r", d->character );
      }

    if ( chance ( 60 ) )
    {
     happymomentbonus = number_range ( 1, 150 );
     d->character->platinum += happymomentbonus;
     printf_to_char ( d->character, "Your {Yplatinum{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Yplatinum{x this time.\n\r", d->character );

    if ( chance ( 30 ) )
    {
     happymomentbonus = number_range ( 1, 2 );
     d->character->questpoints += happymomentbonus;
     printf_to_char ( d->character, "Your {Yiqp{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Yiqps{x this time.\n\r", d->character );

    if ( chance ( 40 ) )
    {
     happymomentbonus = number_range ( 10, 350 );
     d->character->pcdata->questpoints += happymomentbonus;
     printf_to_char ( d->character, "Your {Yaqp{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Yaqps{x this time.\n\r", d->character );

    if ( chance ( 10 ) )
    {
     happymomentbonus = number_range ( 1, 20 );
     d->character->max_hit += happymomentbonus;
     printf_to_char ( d->character, "Your {Yhp{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Yhps{x this time.\n\r", d->character );

    if ( chance ( 12 ) )
    {
     happymomentbonus = number_range ( 1, 30 );
     d->character->max_mana += happymomentbonus;
     printf_to_char ( d->character, "Your {Ymana{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Ymana{x this time.\n\r", d->character );

    if ( chance ( 18 ) )
    {
     happymomentbonus = number_range ( 1, 40 );
     d->character->max_move += happymomentbonus;
     printf_to_char ( d->character, "Your {Ymove{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Ymoves{x this time.\n\r", d->character );

    if ( chance ( 35 ) )
    {
     happymomentbonus = number_range ( 1, 20 );
     d->character->practice += happymomentbonus;
     printf_to_char ( d->character, "Your {Ypractice{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Ypractices{x this time.\n\r", d->character );

    if ( chance ( 35 ) )
    {
     happymomentbonus = number_range ( 1, 5 );
     d->character->train += happymomentbonus;
     printf_to_char ( d->character, "Your {Ytrain{x amount increases by {r({R%ld{r){x!!\n\r", happymomentbonus );
    }
    else send_to_char ( "Sorry, you didn't get any {Ytrains{x this time.\n\r", d->character );

    d->character->hit = d->character->max_hit;
    d->character->mana = d->character->max_mana;
    d->character->move = d->character->max_move;
    send_to_char ( "{*The gods have restored you!{x\n\r", d->character );
   }
 }
}
